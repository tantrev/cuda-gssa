#include "hip/hip_runtime.h"
//Gillespie's Direct Stochastic Simulation Algorithm Program
//Parallel NVIDIA GPU Simulation Code
//Final Project for BIOEN 6760, Modeling and Analysis of Biological Networks
//Trevor James Tanner
//Copyright 2013-2015

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>

//Error checking code for CUDA-related functions
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//Rudimentary version of a Hillis-Steele Scan
__global__ void scan(float* inputArray, int n)
{
	extern __shared__ float sdata[];

	int myId = threadIdx.x + blockDim.x*blockIdx.x;
	int tid = threadIdx.x;

	sdata[tid] = inputArray[myId];
	__syncthreads();

	for (int i = 1; i < n; i *= 2)
	{
		if (tid >= i)
		{
			sdata[tid] += sdata[tid - i];
		}
		__syncthreads();
	}

	inputArray[myId] = sdata[tid];

}

//Binary Search Tree - Upper Bound Search
__host__ __device__ int findTarget(float* inputArray, int startingIndex, int endingIndex, float targetValue)
{
	int length = endingIndex - startingIndex;
	if (length > 1)
	{
		int leftSearchIndex = startingIndex + length / 2 + length % 2;
		int rightSearchIndex = endingIndex;
		float leftSearchValue = inputArray[leftSearchIndex];
		float rightSearchValue = inputArray[rightSearchIndex];
		if (leftSearchValue >= targetValue)
		{
			return findTarget(inputArray, startingIndex, leftSearchIndex, targetValue);
		}
		else if (rightSearchValue >= targetValue)
		{
			return findTarget(inputArray, leftSearchIndex + 1, rightSearchIndex, targetValue);
		}
		else
		{
			return -1;
		}
	}
	else if (inputArray[startingIndex] >= targetValue)
	{
		return startingIndex;
	}
	else if (inputArray[endingIndex] >= targetValue)
	{
		return endingIndex;
	}
	else
	{
		return -1;
	}
}

//Initiates Random States for NVIDIA's Random Number Generator (cuRAND)
__global__ void initStates(hiprandState* globalStateArray, int numTrajectories)
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	while (tId < numTrajectories)
	{
		hiprand_init((unsigned long long)clock(), tId, 0, &globalStateArray[tId]);
		tId += blockDim.x * gridDim.x;
	}
}


int* get2DIntArray(int arraySizeX, int arraySizeY)
{
	int *returnArray = (int*)malloc(arraySizeX*arraySizeY*sizeof(int));
	return returnArray;
}

int** get2DIntArrayOLD(int arraySizeX, int arraySizeY)
{
	int ** returnArray = (int**)malloc(arraySizeX*sizeof(int*));
	for (int i = 0; i < arraySizeX; ++i)
	{
		returnArray[i] = (int*)malloc(sizeof(int)*arraySizeY);
	}
	return returnArray;
}

//Generates random network for simulation
int** getRandom2DIntArrayOLD(int arraySizeX, int arraySizeY, int inputNumSpecies)
{
	int ** returnArray = get2DIntArrayOLD(arraySizeX, arraySizeY);
	for (int i = 0; i < arraySizeX; ++i)
	{
		returnArray[i][0] = rand() % 3; //reactionType
		if (returnArray[i][0] == 0)
		{
			returnArray[i][5] = -1;
			returnArray[i][6] = 0;
			returnArray[i][7] = 1;
			returnArray[i][8] = 0;
			returnArray[i][1] = rand() % inputNumSpecies; //reactantIndex1
			returnArray[i][2] = 0; //reactantIndex2
			returnArray[i][3] = rand() % inputNumSpecies; //productIndex1
			returnArray[i][4] = 0; //productIndex2
		}
		else if (returnArray[i][0] == 1)
		{
			returnArray[i][5] = -1;
			returnArray[i][6] = -1;
			returnArray[i][7] = 1;
			returnArray[i][8] = 0;
			returnArray[i][1] = rand() % inputNumSpecies; //reactantIndex1
			returnArray[i][2] = rand() % inputNumSpecies; //reactantIndex2
			returnArray[i][3] = rand() % inputNumSpecies; //productIndex1
			returnArray[i][4] = 0; //productIndex2
		}
		else
		{
			returnArray[i][5] = -2;
			returnArray[i][6] = 0;
			returnArray[i][7] = 1;
			returnArray[i][8] = 0;
			returnArray[i][1] = rand() % inputNumSpecies; //reactantIndex1
			returnArray[i][2] = 0; //reactantIndex2
			returnArray[i][3] = rand() % inputNumSpecies; //productIndex1
			returnArray[i][4] = 0; //productIndex2
		}
	}
	return returnArray;
}

void free2DArray(int** inputArray, int arraySizeX)
{
	for (int i = 0; i < arraySizeX; ++i)
	{
		free(inputArray[i]);
	}
	free(inputArray);
}
int * getRandomIntArray(int inputSize, int maxSize)
{
	int* r = (int *)malloc(sizeof(int)*inputSize);
	int i;

	for (i = 0; i < inputSize; ++i)
	{
		r[i] = rand() % maxSize;
	}

	return r;
}

float * getRandomFloatArray(int inputSize)
{
	float* r = (float *)malloc(sizeof(float)*inputSize);
	int i;

	for (i = 0; i < inputSize; ++i)
	{
		r[i] = (float)rand() / float(RAND_MAX);
	}

	return r;
}

void calculatePropensities(float* inputPropensityArray, int* inputSpeciesArray, float* inputKeffArray, int* inputReactantMatrix, int inputReactantMatrixWidth, int inputNumReactants)
{
	for (int i = 0; i < inputNumReactants; i++)
	{
		int reactantType = inputReactantMatrix[i*inputReactantMatrixWidth + 0];
		if (reactantType == 0)
		{
			inputPropensityArray[i] = inputKeffArray[i] * inputSpeciesArray[inputReactantMatrix[i*inputReactantMatrixWidth + 1]];
		}
		else if (reactantType == 1)
		{
			inputPropensityArray[i] = inputKeffArray[i] * inputSpeciesArray[inputReactantMatrix[i*inputReactantMatrixWidth + 1]] * inputSpeciesArray[inputReactantMatrix[i*inputReactantMatrixWidth + 2]];
		}
		else
		{
			inputPropensityArray[i] = inputKeffArray[i] * inputSpeciesArray[inputReactantMatrix[i*inputReactantMatrixWidth + 1]] * (inputSpeciesArray[inputReactantMatrix[i*inputReactantMatrixWidth + 1]] - 1) / 2;
		}
	}
}

__global__ void calculatePropensitiesCUDAv2(float* inputPropensityArray, int* inputSpeciesArray, float* inputKeffArray, int* inputReactantMatrix, int inputReactantMatrixWidth, int inputNumSubReactants, int inputTotalNumReactants, int inputNumSubSpecies)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < inputTotalNumReactants)
	{
		int scaledReactantIndex = tid % inputNumSubReactants;

		int scaledSpeciesFactor = tid / inputNumSubReactants;

		int reactantType = inputReactantMatrix[scaledReactantIndex*inputReactantMatrixWidth + 0];

		if (reactantType == 0)
		{
			inputPropensityArray[tid] = inputKeffArray[scaledReactantIndex] * inputSpeciesArray[(scaledSpeciesFactor*inputNumSubSpecies) + inputReactantMatrix[scaledReactantIndex*inputReactantMatrixWidth + 1]];
		}
		else if (reactantType == 1)
		{
			inputPropensityArray[tid] = inputKeffArray[scaledReactantIndex] * inputSpeciesArray[(scaledSpeciesFactor*inputNumSubSpecies) + inputReactantMatrix[scaledReactantIndex*inputReactantMatrixWidth + 1]] * inputSpeciesArray[(scaledSpeciesFactor*inputNumSubSpecies) + inputReactantMatrix[scaledReactantIndex*inputReactantMatrixWidth + 2]];
		}
		else
		{
			inputPropensityArray[tid] = inputKeffArray[scaledReactantIndex] * inputSpeciesArray[(scaledSpeciesFactor*inputNumSubSpecies) + inputReactantMatrix[scaledReactantIndex*inputReactantMatrixWidth + 1]] * (inputSpeciesArray[(scaledSpeciesFactor*inputNumSubSpecies) + inputReactantMatrix[scaledReactantIndex*inputReactantMatrixWidth + 1]] - 1) / 2;
		}
	}
}

void sumPropensities(float *inputPropensityArray, float *inputSummedPropensityArray, int inputNumReactions)
{
	for (int i = 0; i < inputNumReactions; i++)
	{
		if (i > 0)
		{
			inputSummedPropensityArray[i] = inputSummedPropensityArray[i - 1] + inputPropensityArray[i];
		}
		else
		{
			inputSummedPropensityArray[i] = inputPropensityArray[i];
		}
	}
}

typedef struct tauReactantIndex tauReactantIndex;
struct tauReactantIndex
{
	float tau;
	int reactantIndex;
};

typedef struct inputArrays inputArrays;
struct inputArrays
{
	int* speciesArray;
	float* parameterArray;
	int* reactionMatrix;
	int numSpecies;
	int numReactions;
};

inputArrays readInputFiles()
{
	//Read Species File
	FILE *speciesFile;
	char *mode = "r";
	speciesFile = fopen("speciesArray.txt", mode);

	if (speciesFile == NULL) {
		fprintf(stderr, "Can't open species file!\n");
	}

	const size_t line_size = 300;
	char* line = (char*)malloc(line_size);

	fgets(line, line_size, speciesFile);
	int numSpecies;
	sscanf(line, "# %i rows", &numSpecies);
	int* speciesArray = (int*)malloc(numSpecies*sizeof(int));

	int currentSpecieNumber;
	for (int i = 0; i < numSpecies; i++)
	{
		fgets(line, line_size, speciesFile);
		sscanf(line, "%i", &currentSpecieNumber);
		speciesArray[i] = currentSpecieNumber;
	}

	//Read Parameter File
	FILE *parameterFile;
	parameterFile = fopen("parameterArray.txt", mode);

	if (parameterFile == NULL) {
		fprintf(stderr, "Can't open parameter file!\n");
	}

	int numParameters;
	fgets(line, line_size, parameterFile);
	sscanf(line, "# %i rows", &numParameters);

	float* parameterArray = (float*)malloc(numParameters*sizeof(float));
	float currentParameterValue;
	for (int i = 0; i < numParameters; i++)
	{
		fgets(line, line_size, parameterFile);
		sscanf(line, "%e", &currentParameterValue);
		parameterArray[i] = currentParameterValue;
	}

	//Read ReactionMatrix File
	FILE *reactionMatrixFile;
	reactionMatrixFile = fopen("reactionMatrix.txt", mode);

	if (parameterFile == NULL) {
		fprintf(stderr, "Can't open reaction matrix file!\n");
	}

	int numReactions;
	fgets(line, line_size, reactionMatrixFile);
	sscanf(line, "# %i rows", &numReactions);

	int* reactionMatrixArray = (int*)malloc(numReactions * 9 * sizeof(int));
	int reactionType, reactantIndex1, reactantIndex2, productIndex1, productIndex2, reactantDelta1, reactantDelta2, productDelta1, productDelta2;
	for (int i = 0; i < numReactions; i++)
	{
		fgets(line, line_size, reactionMatrixFile);
		sscanf(line, "%i %i %i %i %i %i %i %i %i", &reactionType, &reactantIndex1, &reactantIndex2, &productIndex1, &productIndex2, &reactantDelta1, &reactantDelta2, &productDelta1, &productDelta2);
		reactionMatrixArray[i * 9 + 0] = reactionType;
		reactionMatrixArray[i * 9 + 1] = reactantIndex1;
		reactionMatrixArray[i * 9 + 2] = reactantIndex2;
		reactionMatrixArray[i * 9 + 3] = productIndex1;
		reactionMatrixArray[i * 9 + 4] = productIndex2;
		reactionMatrixArray[i * 9 + 5] = reactantDelta1;
		reactionMatrixArray[i * 9 + 6] = reactantDelta2;
		reactionMatrixArray[i * 9 + 7] = productDelta1;
		reactionMatrixArray[i * 9 + 8] = productDelta2;
	}

	fclose(parameterFile); fclose(speciesFile); fclose(reactionMatrixFile);
	inputArrays returnInputArrays = {speciesArray,parameterArray,reactionMatrixArray,numSpecies,numReactions};

	return returnInputArrays;
}


void fireReaction(int *inputReactionMatrix, int inputReactionMatrixWidth, int *inputSpeciesMatrix, int inputReactionIndex)
{
	int reactantIndex1 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 1];
	int reactantIndex2 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 2];
	int reactantIndex3 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 3];
	int reactantIndex4 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 4];

	int reactantDelta1 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 5];
	int reactantDelta2 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 6];
	int reactantDelta3 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 7];
	int reactantDelta4 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 8];

	int end1 = inputSpeciesMatrix[reactantIndex1] + reactantDelta1;
	int end2 = inputSpeciesMatrix[reactantIndex2] + reactantDelta2;
	int end3 = inputSpeciesMatrix[reactantIndex3] + reactantDelta3;
	int end4 = inputSpeciesMatrix[reactantIndex4] + reactantDelta4;

	if ((end1 < 0) || (end2 < 0) || (end3 < 0) || (end4 < 0))
	{
	}
	else
	{
		inputSpeciesMatrix[reactantIndex1] = end1;
		inputSpeciesMatrix[reactantIndex2] = end2;
		inputSpeciesMatrix[reactantIndex3] = end3;
		inputSpeciesMatrix[reactantIndex4] = end4;
	}
}

__device__ void fireReactionCUDA(int *inputReactionMatrix, int inputReactionMatrixWidth, int *inputSpeciesMatrix, int inputReactionIndex)
{
	int reactantIndex1 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 1];
	int reactantIndex2 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 2];
	int reactantIndex3 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 3];
	int reactantIndex4 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 4];

	int reactantDelta1 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 5];
	int reactantDelta2 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 6];
	int reactantDelta3 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 7];
	int reactantDelta4 = inputReactionMatrix[inputReactionIndex*inputReactionMatrixWidth + 8];

	int end1 = inputSpeciesMatrix[reactantIndex1] + reactantDelta1;
	int end2 = inputSpeciesMatrix[reactantIndex2] + reactantDelta2;
	int end3 = inputSpeciesMatrix[reactantIndex3] + reactantDelta3;
	int end4 = inputSpeciesMatrix[reactantIndex4] + reactantDelta4;

	if ((end1 < 0) || (end2 < 0) || (end3 < 0) || (end4<0))
	{
	}
	else
	{
		inputSpeciesMatrix[reactantIndex1] = end1;
		inputSpeciesMatrix[reactantIndex2] = end2;
		inputSpeciesMatrix[reactantIndex3] = end3;
		inputSpeciesMatrix[reactantIndex4] = end4;
	}
}

__device__ void fireReactionCUDAv2(int *inputReactionMatrix, int inputReactionMatrixWidth, int *inputSpeciesMatrix, int inputReactionIndex, int inputNumSubReactants, int inputNumSubSpecies)
{
	int scaledReactantIndex = inputReactionIndex % inputNumSubReactants;

	int scaledSpeciesFactor = inputReactionIndex / inputNumSubReactants;

	int reactantIndex1 = scaledSpeciesFactor*inputNumSubSpecies + inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 1];
	int reactantIndex2 = scaledSpeciesFactor*inputNumSubSpecies + inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 2];
	int reactantIndex3 = scaledSpeciesFactor*inputNumSubSpecies + inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 3];
	int reactantIndex4 = scaledSpeciesFactor*inputNumSubSpecies + inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 4];

	int reactantDelta1 = inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 5];
	int reactantDelta2 = inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 6];
	int reactantDelta3 = inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 7];
	int reactantDelta4 = inputReactionMatrix[scaledReactantIndex*inputReactionMatrixWidth + 8];

	int end1 = inputSpeciesMatrix[reactantIndex1] + reactantDelta1;
	int end2 = inputSpeciesMatrix[reactantIndex2] + reactantDelta2;
	int end3 = inputSpeciesMatrix[reactantIndex3] + reactantDelta3;
	int end4 = inputSpeciesMatrix[reactantIndex4] + reactantDelta4;

	if ((end1 < 0) || (end2 < 0) || (end3 < 0) || (end4 < 0))
	{
		//if the reactions would have caused negative species, do nothing
	}
	else
	{
		inputSpeciesMatrix[reactantIndex1] = end1;
		inputSpeciesMatrix[reactantIndex2] = end2;
		inputSpeciesMatrix[reactantIndex3] = end3;
		inputSpeciesMatrix[reactantIndex4] = end4;
	}
}

__global__ void findTargets(float* inputArray, int numSubElements, int numTrajectories, hiprandState* globalStateArray, int *inputReactionMatrix, int inputReactionMatrixWidth, int *inputSpeciesMatrix, int inputCurrentTimeStep, float* inputReactionFiredMatrix, int inputNumSubSpecies, int inputNumTimeSteps, float* inputArray2)
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tId < numTrajectories)
	{
		int beginIndex = tId*numSubElements;
		int endIndex = beginIndex + numSubElements - 1;

		float z2 = hiprand_uniform(&globalStateArray[tId]);
		float propensitySum = inputArray[endIndex];
		float tau = log10(propensitySum) / z2;
		float findMe = propensitySum*z2;
		int foundReactionIndex = findTarget(inputArray, beginIndex, endIndex, findMe);
		
		fireReactionCUDAv2(inputReactionMatrix, inputReactionMatrixWidth, inputSpeciesMatrix, foundReactionIndex, numSubElements, inputNumSubSpecies);

		inputReactionFiredMatrix[tId*inputNumTimeSteps + inputCurrentTimeStep * 2 + 0] = tau; inputReactionFiredMatrix[tId*inputNumTimeSteps + inputCurrentTimeStep * 2 + 1] = foundReactionIndex;

	}
}

tauReactantIndex findReactionToFire(float *inputSummedPropensityArray, int inputNumReactions)
{
	float propensitySum = inputSummedPropensityArray[inputNumReactions - 1];
	float z2 = (float)rand() / float(RAND_MAX);
	float tau = log10(propensitySum) / z2;
	float findMe = propensitySum*z2;

	float *p = std::upper_bound(inputSummedPropensityArray, inputSummedPropensityArray + inputNumReactions - 1, findMe);
	int reactionIndex = p - inputSummedPropensityArray;
	tauReactantIndex returnMe = { tau, reactionIndex };
	return returnMe;
}

int comparator(const void *p, const void*q)
{
	const int *leftArray = *(const int**)p;
	const int *rightArray = *(const int**)q;

	int leftValue = leftArray[0];
	int rightValue = rightArray[0];

	return leftValue - rightValue;
}

void runCPUSimulation(float* inputKeff, int* inputReactionMatrix, int* inputSpecies, int* inputCalcSpecies, int inputNumReactions, int inputNumTimeSteps, int inputNumSpecies, float* inputPropensityArray, float* inputSummedPropensityArray, float* inputReactantFiredMatrix)
{
	for (int i = 0; i < inputNumTimeSteps; ++i)
	{
		calculatePropensities(inputPropensityArray, inputCalcSpecies, inputKeff, inputReactionMatrix, 9, inputNumReactions);
		sumPropensities(inputPropensityArray, inputSummedPropensityArray, inputNumReactions);
		tauReactantIndex tauReactantObject = findReactionToFire(inputSummedPropensityArray, inputNumReactions);
		inputReactantFiredMatrix[i * 2 + 0] = tauReactantObject.tau; inputReactantFiredMatrix[i * 2 + 1] = tauReactantObject.reactantIndex;
		fireReaction(inputReactionMatrix, 9, inputCalcSpecies, tauReactantObject.reactantIndex);
	}

}

void runGPUSimulationv3(float* inputKeff_CUDA, int* inputReactionMatrix_CUDA, int* inputSpecies_CUDA, int* inputCalcSpecies_CUDA, int inputNumReactions, int inputNumTimeSteps, int inputNumSpecies, float* inputPropensityArray_CUDA, float* inputSummedPropensityArray_CUDA, float* inputReactantFiredMatrix_CUDA, float* inputReactantFiredMatrix_HOST, int inputNumTrajectories, hiprandState* globalStateArray)
{
	int threadsPerBlock = 32;

	for (int j = 0; j < inputNumTimeSteps; ++j)
	{
		calculatePropensitiesCUDAv2 <<<(inputNumTrajectories*inputNumReactions + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >>>(inputPropensityArray_CUDA, inputSpecies_CUDA, inputKeff_CUDA, inputReactionMatrix_CUDA, 9, inputNumReactions, inputNumReactions*inputNumTrajectories, inputNumSpecies);
		scan <<<inputNumTrajectories, inputNumReactions, inputNumReactions*sizeof(float) >>>(inputPropensityArray_CUDA, inputNumReactions);
		findTargets <<<(inputNumTrajectories + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >>>(inputPropensityArray_CUDA, inputNumReactions, inputNumTrajectories, globalStateArray, inputReactionMatrix_CUDA, 9, inputSpecies_CUDA, j, inputReactantFiredMatrix_CUDA, inputNumSpecies, inputNumTimeSteps, inputPropensityArray_CUDA);
	}

	gpuErrchk(hipMemcpy(inputReactantFiredMatrix_HOST, inputReactantFiredMatrix_CUDA, inputNumTrajectories* inputNumTimeSteps * 2 * sizeof(float), hipMemcpyDeviceToHost));

}

__global__ void warmUp()
{
}

int * flatten2DArray(int** input2DArray, int inputSizeX, int inputSizeY)
{
	int * returnArray = get2DIntArray(inputSizeX, inputSizeY);
	for (int i = 0; i < inputSizeX; ++i)
	{
		for (int j = 0; j < inputSizeY; ++j)
		{
			returnArray[i*inputSizeY + j] = input2DArray[i][j];
		}
	}
	return returnArray;
}

void printTimings(bool inputReadFile, int inputNumRandomReactions, int inputNumRandSpecies, int inputNumTimeSteps, int inputNumSimulations)
{
	clock_t begin_CPU, end_CPU, begin_GPU, end_GPU;
	float time_spent_GPU, time_spent_CPU;

	float *kEff;
	int *reactionMatrix;
	int *species;
	inputArrays inputArraysRead;

	int numSimulations = inputNumSimulations;
	int numTimeSteps = inputNumTimeSteps;

	int numReactions;
	int numSpecies;

	if (inputReadFile == true)
	{
		inputArraysRead = readInputFiles();
		numReactions = inputArraysRead.numReactions;
		numSpecies = inputArraysRead.numSpecies;
		reactionMatrix = inputArraysRead.reactionMatrix;
		species = inputArraysRead.speciesArray;
		kEff = inputArraysRead.parameterArray;
	}
	else
	{
		numReactions = inputNumRandomReactions;
		numSpecies = inputNumRandSpecies;
		species = getRandomIntArray(numSpecies, 100);
		kEff = getRandomFloatArray(numReactions);
		int **reactionMatrixOLD = getRandom2DIntArrayOLD(numReactions, 9, numSpecies);
		qsort(reactionMatrixOLD, numReactions, sizeof(int), comparator); //Sort the array to make branch prediction work
		reactionMatrix = flatten2DArray(reactionMatrixOLD, numReactions, 9);
		free2DArray(reactionMatrixOLD, numSpecies);
	}

	printf("readFile:%d numReactions:%i numSpecies:%i numTimeSteps:%i numSimulations:%i\n", inputReadFile, numReactions, numSpecies, numTimeSteps, numSimulations);

	//These guys will always be changing
	int* calcSpecies = (int *)malloc(sizeof(int)*numSpecies);
	std::copy(species, species + numSpecies, calcSpecies);
	float *propensityArray = (float *)malloc(sizeof(float)*numReactions); //initially empty
	float *summedPropensityArray = (float *)malloc(sizeof(float)*numReactions); //initially empty
	//OUTPUT
	float *reactantFiredMatrix = (float *)malloc(numTimeSteps * 2 * sizeof(float)); //column1=time,column2=reactionFired

	//INPUTS SPECIFICALLY FOR GPU SIMULATION (some of the CPU inputs are reused)
	int* species_HOST = (int *)malloc(sizeof(int)*numSpecies*numSimulations);
	int* calcSpecies_HOST = (int *)malloc(sizeof(int)*numSpecies*numSimulations);
	float *propensityArray_HOST = (float *)malloc(sizeof(float)*numReactions*numSimulations);
	float *summedPropensityArray_HOST = (float *)malloc(sizeof(float)*numReactions*numSimulations);
	float* reactantFiredMatrix_HOST = (float *)malloc(numSimulations*numTimeSteps * 2 * sizeof(float));
	for (int l = 0; l < numSimulations; l++)
	{
		for (int k = 0; k < numReactions; k++)
		{
			propensityArray_HOST[l*numReactions + k] = propensityArray[k];
		}
		for (int m = 0; m < numSpecies; m++)
		{
			species_HOST[l*numSpecies + m] = species[m];
			calcSpecies_HOST[l*numSpecies + m] = species[m];
		}
	}

	//CUDA Variable Versions
	float *kEffCUDA;
	int *reactionMatrixCUDA;
	int *speciesCUDA;
	int *calcSpeciesCUDA;
	float *propensityArrayCUDA;
	float *summedPropensityArrayCUDA;
	float *reactantFiredMatrixCUDA;

	//Make Device Pointers
	gpuErrchk(hipMalloc(&reactionMatrixCUDA, numReactions * 9 * sizeof(int)));
	gpuErrchk(hipMalloc(&kEffCUDA, numReactions*sizeof(float)));
	gpuErrchk(hipMalloc(&speciesCUDA, numSimulations*numSpecies*sizeof(int)));
	gpuErrchk(hipMalloc(&calcSpeciesCUDA, numSimulations*numSpecies*sizeof(int)));
	gpuErrchk(hipMalloc(&propensityArrayCUDA, numSimulations*numReactions*sizeof(float)));
	gpuErrchk(hipMalloc(&summedPropensityArrayCUDA, numSimulations*numReactions*sizeof(float)));
	gpuErrchk(hipMalloc(&reactantFiredMatrixCUDA, numSimulations*numTimeSteps * 2 * sizeof(float)));

	//Copy Data to Device
	gpuErrchk(hipMemcpy(reactionMatrixCUDA, reactionMatrix, numReactions * 9 * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(kEffCUDA, kEff, numReactions*sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(speciesCUDA, species_HOST, numSimulations*numSpecies*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(calcSpeciesCUDA, calcSpecies_HOST, numSimulations*numSpecies*sizeof(int), hipMemcpyHostToDevice));

	printf("Starting!\n");
	//GPU Timing
	warmUp << <1, 1 >> >();
	hiprandState* globalStateArrayInput;
	gpuErrchk(hipMalloc(&globalStateArrayInput, numSimulations * sizeof(hiprandState)));
	int threadsPerBlock = 32;
	initStates << <(numSimulations + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock >> >(globalStateArrayInput, numSimulations);

	begin_GPU = clock();
	runGPUSimulationv3(kEffCUDA, reactionMatrixCUDA, speciesCUDA, calcSpeciesCUDA, numReactions, numTimeSteps, numSpecies, propensityArrayCUDA, summedPropensityArrayCUDA, reactantFiredMatrixCUDA, reactantFiredMatrix_HOST, numSimulations, globalStateArrayInput);

	hipDeviceSynchronize();
	end_GPU = clock();

	printf("Ending!\n");

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	time_spent_GPU = (float)(end_GPU - begin_GPU) / CLOCKS_PER_SEC;

	float avg_GPU = (time_spent_GPU) / numSimulations;
	printf("Avg. GPU Simulation Time: %.17g [sim/sec]\n", avg_GPU);
	hipFree(reactionMatrixCUDA); hipFree(kEffCUDA); hipFree(speciesCUDA); hipFree(calcSpeciesCUDA); hipFree(propensityArrayCUDA); hipFree(summedPropensityArrayCUDA); hipFree(reactantFiredMatrixCUDA); hipFree(globalStateArrayInput);
	free(species_HOST); free(calcSpecies_HOST); free(propensityArray_HOST); free(summedPropensityArray_HOST); free(reactantFiredMatrix_HOST);
	hipDeviceSynchronize();
	hipDeviceReset();

	////CPU Timing
	//begin_CPU = clock();

	//for (int j = 0; j < numSimulations; ++j)
	//{
	//	runCPUSimulation(kEff, reactionMatrix, species, calcSpecies, numReactions, numTimeSteps, numSpecies, propensityArray, summedPropensityArray, reactantFiredMatrix);
	//}

	//end_CPU = clock();

	////Clean-up 
	free(kEff); free(species); free(calcSpecies);  free(reactionMatrix); free(propensityArray); free(summedPropensityArray); free(reactantFiredMatrix);

	//time_spent_CPU = (float)(end_CPU - begin_CPU) / CLOCKS_PER_SEC;
	//float avg_CPU = time_spent_CPU / numSimulations;
	//printf("Avg. CPU Simulation Time: %.17g [sim/sec]\n", avg_CPU);
	//printf("CPU/GPU Diff:%.17g\n", avg_CPU / avg_GPU);
}

int main(int argc, char** argv)
{
	printTimings(true, 1024, 1024, 10000, 1000);

	int numSpeciesReactions = 1;
	for (int i = 1; i <= 11; i++)
	{
		numSpeciesReactions *= 2;
		printTimings(false, numSpeciesReactions / 2, numSpeciesReactions / 2, 10000, 1000);
	}

	return 0;
}